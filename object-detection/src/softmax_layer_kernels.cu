#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"


#include "softmax_layer.h"
#include "hip/hip_runtime.h"
#include "blas.h"


__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        int val = input[i+b*n];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        sum += exp(input[i+b*n]/temp-largest/temp);
    }
    sum = (sum != 0) ? largest/temp+log(sum) : largest-100;
    for(i = 0; i < n; ++i){
        output[i+b*n] = exp(input[i+b*n]/temp-sum);
    }
}

void pull_softmax_layer_output(const softmax_layer layer)
{
    cuda_pull_array(layer.output_gpu, layer.output, layer.inputs*layer.batch);
}

void forward_softmax_layer_gpu(const softmax_layer layer, network_state state)
{
    int inputs = layer.inputs / layer.groups;
    int batch = layer.batch * layer.groups;
    forward_softmax_layer_kernel<<<cuda_gridsize(batch), BLOCK>>>(inputs, batch, state.input, layer.temperature, layer.output_gpu);
    check_error(hipPeekAtLastError());
}

void backward_softmax_layer_gpu(const softmax_layer layer, network_state state)
{
    axpy_ongpu(layer.batch*layer.inputs, 1, layer.delta_gpu, 1, state.delta, 1);
}

/* This is if you want softmax w/o log-loss classification. You probably don't.
   int i,j,b;
   for(b = 0; b < layer.batch; ++b){
   for(i = 0; i < layer.inputs; ++i){
   for(j = 0; j < layer.inputs; ++j){
   int d = (i==j);
   layer.jacobian[b*layer.inputs*layer.inputs + i*layer.inputs + j] = 
   layer.output[b*layer.inputs + i] * (d - layer.output[b*layer.inputs + j]);
   }
   }
   }
   for(b = 0; b < layer.batch; ++b){
   int M = layer.inputs;
   int N = 1;
   int K = layer.inputs;
   float *A = layer.jacobian + b*layer.inputs*layer.inputs;
   float *B = layer.delta + b*layer.inputs;
   float *C = delta + b*layer.inputs;
   gemm(0,0,M,N,K,1,A,K,B,N,0,C,N);
   }
 */
